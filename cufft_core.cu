
#include <hip/hip_runtime.h>
#include <cmath>
#include <hipfft/hipfft.h>

void cufft_core_execute(float* Signal_h, int Size, float2* SignalFFT_h) {
    float *Signal_d;
    float2 *SignalFFT_d;
    hipMalloc((void**)&Signal_d, Size*sizeof(float));
    hipMalloc((void**)&SignalFFT_d, (Size/2+1)*sizeof(float2));
    hipMemcpy(Signal_d, Signal_h, Size*sizeof(float), hipMemcpyHostToDevice);
    hipfftHandle Plan;
    hipfftPlan1d(&Plan, Size, HIPFFT_R2C, 1);
    hipfftExecR2C(Plan, (hipfftReal*)Signal_d, (hipfftComplex*)SignalFFT_d);
    hipMemcpy(SignalFFT_h, SignalFFT_d, (Size/2+1)*sizeof(float2), hipMemcpyDeviceToHost);
    hipFree(Signal_d);
    hipFree(SignalFFT_d);
}